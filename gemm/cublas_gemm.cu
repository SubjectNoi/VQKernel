
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdio>
#include <iostream>

using namespace std;

#define CHECK_CUBLAS(Expr) { \
    int err = (Expr); \
    if (err != 0) { \
        printf("cuBLAS error %d at line %d\n", err, __LINE__); \
    } \
}

void gemm(hipblasHandle_t handle,
          int m,
          int n,
          int k,
          const void *alpha,
          const void *beta,
          hipDataType input_type,
          const void *A,
          const void *B,
          hipDataType output_type,
          void *C,
#if __CUDACC_VER_MAJOR__ >= 11
          cublasComputeType_t compute_type,
#else
          hipDataType compute_type,
#endif
          hipblasGemmAlgo_t algo) {
    CHECK_CUBLAS(hipblasGemmEx(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
        alpha, B, input_type, n, A, input_type, k,
        beta, C, output_type, n, compute_type, algo));
}

int main(int args, char* argv[]) {

    // On RTX 6000 Ada
    // nvcc -o cublas_gemm cublas_gemm.cu -lcublas -code=sm_89 -arch=compute_89
    // m,n,k = 49152, 128, 64, mps=10%, latency=0.049252ms, 
    // at the mean time, juno latency = 59us
    // They can perfectly pipelined
    int m = std::atoi(argv[1]);
    int n = std::atoi(argv[2]);
    int k = std::atoi(argv[3]);

    __half alpha = __float2half(1.0f);
    __half beta = __float2half(0.0f);

    hipDataType input_type = HIP_R_16F;
    hipDataType output_type = HIP_R_16F;
#if __CUDACC_VER_MAJOR__ >= 11
    cublasComputeType_t compute_type = CUBLAS_COMPUTE_16F;
#else
    hipDataType compute_type = HIP_R_16F;
#endif
    hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;

    int iter = 100;

    void *A, *B, *C;
    hipMalloc(&A, m * k * sizeof(__half));
    hipMalloc(&B, k * n * sizeof(__half));
    hipMalloc(&C, m * n * sizeof(__half));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // warmup
    gemm(handle, m, n, k, &alpha, &beta, input_type, A, B,
         output_type, C, compute_type, algo);

    hipEventRecord(start);
    for (int i = 0; i < iter; ++i) {
        gemm(handle, m, n, k, &alpha, &beta, input_type, A, B,
             output_type, C, compute_type, algo);
    }
    hipEventRecord(stop);

    float time_ms = 0.f;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_ms, start, stop);

    long ops = (long)m * n * k * 2;
    double gops = ((double)ops / 1e9) / ((double)time_ms / iter / 1e3) / 1e3;
    printf("CBLAS - M : %d, N : %d, K : %d, %f ms, %f Tflops\n", m, n, k, (time_ms/iter), gops);

    hipFree(A);
    hipFree(B);
    hipFree(C);
}